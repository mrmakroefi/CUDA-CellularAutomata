#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <algorithm>
#include <chrono>
#include <vector>
#include <queue>
#include <ppl.h>

#define dllexp __declspec(dllexport)
#define uint unsigned int
#define byte uint8_t
#define ushort unsigned short
#define vector std::vector

class Room {
public:
	vector<uint> tiles;
	vector<uint> edgesTiles;
	vector<Room*> connectedRooms;
	uint roomSize;
	bool isAccessibleFromMainRoom;
	bool isMainRoom;

	Room() {

	}

	Room(vector<uint> roomTiles, byte* map, uint width, uint height) {
		isAccessibleFromMainRoom = false;
		isMainRoom = false;
		tiles = roomTiles;
		roomSize = roomTiles.size();

		// get the edges tiles of the room
		for each (uint tile in tiles)
		{
			uint x = tile % width;
			uint y = (tile - x) / width;

			int xdown = x - 1, xup = x + 1;
			int ydown = y - 1, yup = y + 1;

			for (int X = xdown; X <= xup; X++) {
				for (int Y = ydown; Y <= yup; Y++) {
					if (X == x || Y == y) {
						if (map[Y * width + X] == 1) {
							edgesTiles.push_back(tile);
						}
					}
				}
			}
		}
	}

	void SetAccessibleFromMainRoom() {
		if (!isAccessibleFromMainRoom) {
			isAccessibleFromMainRoom = true;
			for (size_t i = 0; i < connectedRooms.size(); i++)
			{
				connectedRooms.at(i)->SetAccessibleFromMainRoom();
			}
		}
	}

	// connect two rooms
	static void ConnectRooms(Room *roomA, Room *roomB) {
		if (roomA->isAccessibleFromMainRoom) {
			roomB->SetAccessibleFromMainRoom();
		}
		else if (roomB->isAccessibleFromMainRoom) {
			roomA->SetAccessibleFromMainRoom();
		}
		roomA->connectedRooms.push_back(roomB);
		roomB->connectedRooms.push_back(roomA);
	}

	// check the connection between two rooms
	bool isConnected(Room *otherRoom) {
		bool found = false;
		for (size_t i = 0; i < connectedRooms.size(); i++)
		{
			if (connectedRooms.at(i)->isSame(otherRoom)) {
				found = true;
				break;
			}
		}
		return found;
	}

	bool isSame(const Room* room) {
		bool flag = this == room;
		/*printf("checking %p -> %i with %p -> %i\n", this, roomSize, &room, room->roomSize);
		if (flag) {
			printf("there's same room detected\n");
		}*/
		return flag;
	}
};

extern "C" {

	dllexp char* getDeviceName();
	dllexp void GPU_simulateCA(byte* data, ushort width, ushort height, byte chanceToAlive, byte starvationLimit, byte birthLimit, int seed, byte iterationsCount, ushort threadsCount);
	dllexp void CPU_simulateCA(byte* data, ushort width, ushort height, byte chanceToAlive, byte starvationLimit, byte birthLimit, int seed, byte iterationsCount);
	dllexp void CPU_floodFill(byte *data, uint width, uint height, uint wallThreshold, uint floorThreshold);

	dllexp void CPU_initMap(byte* data, uint size, byte chanceToAlive, uint seed);
}

vector<uint> GetRegionTiles(byte *data, uint width, uint height, byte *mapFlags, uint id);
vector<vector<uint>> GetRegions(byte *data, uint width, uint height, byte tileType);
void ConnectClosestRooms(vector<Room*> allRooms, byte* data, uint width, uint height, bool forceAccessibility = false);
void CreatePassage(byte* data, uint width, uint height, Room *roomA, Room *roomB, uint tileA, uint tileB);
void DrawCircle(uint id, int r, byte* data, uint width, uint height);
vector<uint> GetLine(uint from, uint to, uint width);

int sign(int x) {
	return (x > 0) ? 1 : ((x < 0) ? -1 : 0);
}

// init map, fill map with 0 and 1
/*__global__ void GPU_InitMap(byte* output, uint size, uint aliveChance, uint seed) {

for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < size; cellId += blockDim.x * gridDim.x) {
hiprandState state;
hiprand_init(seed, cellId, 0, &state);
uint value = (uint)(hiprand_uniform(&state) * 100) <= aliveChance ? 1 : 0;
output[cellId] = value;
}
}*/

__global__ void simulateCA(byte* data, uint width, uint height, byte* dataBuffer, byte starvationLimit, byte birthLimit) {
	uint worldSize = width * height;

	// initiate threads
	for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {

		// define x and y coord
		uint x = cellId % width;
		uint y = (cellId - x) / width;

		if (x == 0 || x == width - 1 || y == 0 || y == height - 1) {
			dataBuffer[cellId] = 1;
		}
		else {
			// get the neighboors
			uint aliveNeighboors = 0;
			for (int j = -1; j < 2; j++) {
				for (int i = -1; i < 2; i++) {
					// ignore the current checked cell
					if (j == 0 && i == 0) continue;

					// get neighboors cell
					int xNeighbour = x + j;
					int yNeighbour = y + i;

					// if out of map
					if (xNeighbour < 0 || yNeighbour < 0 || xNeighbour >= width || yNeighbour >= height) {
						aliveNeighboors++;
					}
					else if (data[yNeighbour * width + xNeighbour] == 1) {
						aliveNeighboors++;
					}

				}
			}

			if (data[cellId] == 1) {
				dataBuffer[cellId] = aliveNeighboors < starvationLimit ? 0 : 1;
			}
			else {
				dataBuffer[cellId] = aliveNeighboors > birthLimit ? 1 : 0;
			}
		}

	}
}

void deleteThis(int* ptr) {
	delete ptr;
}

char* getDeviceName() {

	hipDeviceProp_t device;
	hipGetDeviceProperties(&device, 0);
	char* nameLabel = new char[256];
	std::strcpy(nameLabel, device.name);
	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	std::cout << "using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
	std::cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << std::endl;
	return nameLabel;
}

void GPU_simulateCA(byte* data, ushort width, ushort height, byte chanceToAlive, byte starvationLimit, byte birthLimit, int seed, byte iterationsCount, ushort threadsCount) {
	uint worldSize = width * height;
	byte* dev_data = new byte[worldSize];
	byte* dev_dataBuffer = new byte[worldSize];

	hipMalloc((void**)&dev_data, sizeof(byte) * worldSize);
	hipMalloc((void**)&dev_dataBuffer, sizeof(byte) * worldSize);

	// configure thread per block
	size_t reqBlocksCount = worldSize / threadsCount;
	ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

	hipError_t cudaStatus;

	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	CPU_initMap(data, worldSize, chanceToAlive, seed);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Time to init world : " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << std::endl;

	hipMemcpy(dev_data, data, sizeof(byte) * worldSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_dataBuffer, data, sizeof(byte) * worldSize, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	// start CA iterations
	for (byte i = 0; i < iterationsCount; i++) {
		simulateCA << <blocksCount, threadsCount >> > (dev_data, width, height, dev_dataBuffer, starvationLimit, birthLimit);

		std::swap(dev_data, dev_dataBuffer);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time to simulate world :   %3.1f ms \n", time);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "simulateCA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipMemcpy(data, dev_data, sizeof(byte) * worldSize, hipMemcpyDeviceToHost);


Error:
	hipFree(dev_data);
	hipFree(dev_dataBuffer);
}

void CPU_initMap(byte* data, uint size, byte chanceToAlive, uint seed) {
	srand(seed);
	for (uint id = 0; id < size; id++) {
		uint value = (rand() % 100) + 1 <= chanceToAlive ? 1 : 0;
		data[id] = value;
	}
}

void CPU_simulateCA(byte* data, ushort width, ushort height, byte chanceToAlive, byte starvationLimit, byte birthLimit, int seed, byte iterationsCount) {
	uint worldSize = width * height;
	byte* dataTemp = new byte[worldSize];
	byte* dataBuffer = new byte[worldSize];

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	CPU_initMap(dataTemp, worldSize, chanceToAlive, seed);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	std::cout << "Time to init world : " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << std::endl;

	std::memcpy(dataBuffer, dataTemp, sizeof(byte) * worldSize);

	begin = std::chrono::steady_clock::now();

	for (byte p = 0; p < iterationsCount; p++) {
		for (uint id = 0; id < worldSize; id++) {
			// define x and y coord
			uint x = id % width;
			uint y = (id - x) / width;

			if (x == 0 || x == width - 1 || y == 0 || y == height - 1) {
				dataBuffer[id] = 1;
			}
			else {
				// get the neighboors
				uint aliveNeighboors = 0;
				for (int j = -1; j < 2; j++) {
					for (int i = -1; i < 2; i++) {
						// ignore the current checked cell
						if (j == 0 && i == 0) continue;

						// get neighboors cell
						int xNeighbour = x + j;
						int yNeighbour = y + i;

						// if out of map
						if (xNeighbour < 0 || yNeighbour < 0 || xNeighbour >= width || yNeighbour >= height) {
							//printf("------ Out of map (%i, %i)\n", xNeighbour, yNeighbour);
							aliveNeighboors++;
						}
						else if (dataTemp[yNeighbour * width + xNeighbour] == 1) {
							//printf("------ Alive neighboor (%i, %i)\n", xNeighbour, yNeighbour);
							aliveNeighboors++;
						}
					}
				}

				if (dataTemp[id] == 1) {
					dataBuffer[id] = aliveNeighboors < starvationLimit ? 0 : 1;
				}
				else {
					dataBuffer[id] = aliveNeighboors > birthLimit ? 1 : 0;
				}
			}
		}

		std::swap(dataBuffer, dataTemp);
	}
	end = std::chrono::steady_clock::now();

	std::cout << "Time to simulate world : " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << std::endl;

	std::memcpy(data, dataTemp, sizeof(byte) * worldSize);
	delete dataBuffer;
	delete dataTemp;
}

vector<uint> GetRegionTiles(byte *data, uint width, uint height, byte *mapFlags, uint id) {
	vector<uint> tiles;
	byte tileType = data[id];
	std::queue<uint> q;
	q.push(id);
	mapFlags[id] = 1;

	while (!q.empty()) {
		uint tile = q.front();
		q.pop();
		tiles.push_back(tile);

		uint x = tile % width,
			y = (tile - x) / width;

		int xdown = x - 1, xup = x + 1;
		int ydown = y - 1, yup = y + 1;

		for (int X = xdown; X <= xup; X++) {
			for (int Y = ydown; Y <= yup; Y++) {
				if ((X == x || Y == y)) {
					if ((X >= 0 && Y >= 0 && X < width && Y < height)) {
						uint neighbourID = (Y * width) + X;
						if (mapFlags[neighbourID] == 0 && data[neighbourID] == tileType) {
							mapFlags[neighbourID] = 1;
							q.push(neighbourID);
						}
					}
				}
			}
		}
	}

	return tiles;
}

vector<vector<uint>> GetRegions(byte *data, uint width, uint height, byte tileType) {
	uint size = width * height;
	vector<vector<uint>> regions;
	byte *mapFlags = new byte[size];
	size_t i;

	for (i = 0; i < size; i++) {
		mapFlags[i] = 0;
	}

	for (i = 0; i < size; i++) {
		if (mapFlags[i] == 0 && data[i] == tileType) {
			vector<uint> newRegion;
			newRegion = GetRegionTiles(data, width, height, mapFlags, i);
			regions.push_back(newRegion);
		}
	}

	delete mapFlags;
	return regions;
}

void CPU_floodFill(byte *data, uint width, uint height, uint wallThreshold, uint floorThreshold) {
	size_t i;
	std::chrono::steady_clock::time_point begin;
	std::chrono::steady_clock::time_point end;

	// remove small wall
	if (wallThreshold > 0) {
		begin = std::chrono::steady_clock::now();
		vector<vector<uint>> wallRegions = GetRegions(data, width, height, 1);
		for (i = 0; i < wallRegions.size(); i++)
		{
			if (wallRegions.at(i).size() < wallThreshold) {
				for (int j = 0; j < wallRegions.at(i).size(); j++)
				{
					data[wallRegions.at(i).at(j)] = 0;
				}
			}
		}
		end = std::chrono::steady_clock::now();
		std::cout << "Took " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << " to remove annoying wall(s)." << std::endl;
	}

	// fill small room
	if (floorThreshold > 0) {
		begin = std::chrono::steady_clock::now();
		vector<vector<uint>> floorRegions = GetRegions(data, width, height, 0);
		vector<Room*> survivedRooms;
		for (i = 0; i < floorRegions.size(); i++)
		{
			if (floorRegions.at(i).size() < floorThreshold) {
				for (int j = 0; j < floorRegions.at(i).size(); j++)
				{
					data[floorRegions.at(i).at(j)] = 1;
				}
			}
			else {
				Room* room = new Room(floorRegions.at(i), data, width, height);
				survivedRooms.push_back(room);
			}
		}
		end = std::chrono::steady_clock::now();
		std::cout << "Took " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << " to fill useless room(s)." << std::endl;

		begin = std::chrono::steady_clock::now();
		// descending order
		std::sort(survivedRooms.begin(), survivedRooms.end(), [](Room* a, Room* b) {
			return a->roomSize > b->roomSize;
		});

		survivedRooms.at(0)->isMainRoom = true;
		survivedRooms.at(0)->isAccessibleFromMainRoom = true;

		ConnectClosestRooms(survivedRooms, data, width, height);
		end = std::chrono::steady_clock::now();
		std::cout << "Took " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000.0 << " ms" << " to connect room(s)." << std::endl;

	}
}

void ConnectClosestRooms(vector<Room*> allRooms, byte* data, uint width, uint height, bool forceAccessibility) {
	vector<Room*> roomListA, roomListB;

	if (forceAccessibility) {
		for (int i = 0; i < allRooms.size(); i++) {
			bool flag = allRooms.at(i)->isAccessibleFromMainRoom;

			if (flag) {
				roomListB.push_back(allRooms.at(i));
			}
			else {
				roomListA.push_back(allRooms.at(i));
			}
		}
	}
	else {
		for (int i = 0; i < allRooms.size(); i++) {
			roomListA.push_back(allRooms.at(i));
			roomListB.push_back(allRooms.at(i));
		}
	}

	int bestDistance = 0;
	uint bestTileA, bestTileB;
	Room *bestRoomA, *bestRoomB;
	bool possibleConnectionFound = false;


	for (int i = 0; i < roomListA.size(); i++) {
		if (!forceAccessibility) {
			possibleConnectionFound = false;
			if (roomListA.at(i)->connectedRooms.size() > 0) {
				continue;
			}
		}
		for (int j = 0; j < roomListB.size(); j++) {
			if (roomListA.at(i)->isSame(roomListB.at(j)) || roomListA.at(i)->isConnected(roomListB.at(j))) {
				continue;
			}

			for (int tileA = 0; tileA < roomListA.at(i)->edgesTiles.size(); tileA+=3) {
				for (int tileB = 0; tileB < roomListB.at(j)->edgesTiles.size(); tileB+=3) {
					uint tileIdA = roomListA.at(i)->edgesTiles.at(tileA);
					uint tileIdB = roomListB.at(j)->edgesTiles.at(tileB);

					int xA = tileIdA % width;
					int yA = (tileIdA - xA) / width;
					int xB = tileIdB % width;
					int yB = (tileIdB - xB) / width;

					int distance = std::pow(xA - xB, 2) + std::pow(yA - yB, 2);
					
					if (distance < bestDistance || !possibleConnectionFound) {
						bestDistance = distance;
						possibleConnectionFound = true;
						bestTileA = tileIdA;
						bestTileB = tileIdB;
						bestRoomA = roomListA.at(i);
						bestRoomB = roomListB.at(j);
					}
				}
			}
		}
		if (possibleConnectionFound && !forceAccessibility) {
			CreatePassage(data, width, height, bestRoomA, bestRoomB, bestTileA, bestTileB);
		}
	}

	if (possibleConnectionFound && forceAccessibility) {
		CreatePassage(data, width, height, bestRoomA, bestRoomB, bestTileA, bestTileB);
		ConnectClosestRooms(allRooms, data, width, height, true);
	}

	if (!forceAccessibility) {
		ConnectClosestRooms(allRooms, data, width, height, true);
	}
}

void CreatePassage(byte* data, uint width, uint height, Room *roomA, Room *roomB, uint tileA, uint tileB) {
	Room::ConnectRooms(roomA, roomB);

	vector<uint> line = GetLine(tileA, tileB, width);
	for each(uint c in line) {
		DrawCircle(c, 1, data, width, height);
	}
}

void DrawCircle(uint id, int r, byte* data, uint width, uint height) {
	for (int x = -r; x <= r; x++) {
		for (int y = -r; y <= r; y++) {
			if (x*x + y*y <= r*r) {
				int drawX = (id % width) + x;
				int drawY = ((id - id%width) / width) + y;

				if (drawX >= 0 && drawY >= 0 && drawX < width && drawY < height) {
					data[drawY * width + drawX] = 0;
				}
			}
		}
	}
}

vector<uint> GetLine(uint from, uint to, uint width) {
	vector<uint> line;

	uint x = from % width,
		y = (from - x) / width;
	uint xTo = to % width,
		yTo = (to - xTo) / width;

	bool inverted = false;
	int dx = xTo - x;
	int dy = yTo - y;

	int step = sign(dx);
	int gradientStep = sign(dy);

	int longest = abs(dx);
	int shortest = abs(dy);

	if (longest < shortest) {
		inverted = true;
		longest = abs(dy);
		shortest = abs(dx);

		step = sign(dy);
		gradientStep = sign(dx);
	}

	int gradientAccumulation = longest / 2;
	for (int i = 0; i < longest; i++) {
		line.push_back(y * width + x);
		gradientAccumulation += shortest;

		if (inverted) {
			y += step;
		}
		else {
			x += step;
		}

		if (gradientAccumulation >= longest) {
			if (inverted) {
				x += gradientStep;
			}
			else {
				y += gradientStep;
			}
			gradientAccumulation -= longest;
		}
	}
	return line;
}
